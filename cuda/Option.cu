#include "hip/hip_runtime.h"
#include "Option.cuh"
#include <math.h>

// Função normal cumulativa (aproximação rápida)
__device__ float norm_cdf(float x) {
    return 0.5f * erfcf(-x * M_SQRT1_2);
}

// Kernel CUDA para precificação Black-Scholes de várias opções
__global__ void priceOptionsBlackScholes(const Option* options, float* prices, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Option opt = options[idx];

    float S = opt.S;
    float K = opt.K;
    float T = opt.T;
    float r = opt.r;
    float sigma = opt.sigma;
    int isCall = opt.isCall;

    float d1 = (logf(S/K) + (r + 0.5f*sigma*sigma)*T) / (sigma*sqrtf(T));
    float d2 = d1 - sigma*sqrtf(T);

    float call = S * norm_cdf(d1) - K * expf(-r*T) * norm_cdf(d2);
    float put  = K * expf(-r*T) * norm_cdf(-d2) - S * norm_cdf(-d1);

    prices[idx] = isCall ? call : put;
}
